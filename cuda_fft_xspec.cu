#include "hip/hip_runtime.h"
//
// FFT using CuFFT
//
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <string.h>
#include <math.h>
#include </usr/local/cuda-8.0/samples/common/inc/timer.h>
#include "cuda_polaris.inc"

#define SCALEFACT 2.0/(NFFT* NsegSec* PARTNUM)

extern int gaussBit(int, unsigned int *, double *, double *);
extern int k5utc(unsigned char *,	struct SHM_PARAM *);
extern int segment_offset(struct SHM_PARAM *,	int *);
extern int fileRecOpen(struct SHM_PARAM *, int, int, char *, char *, FILE **);
extern int bitDist4(int, unsigned char *, unsigned int *);
extern int bitDist8(int, unsigned char *, unsigned int *);


int main(int argc, char **argv)
{
    int           shrd_param_id;          // Shared Memory ID
    int           index;                  // General Index
    int           part_index;             // First and Last Part
    int           seg_index;              // Index for Segment
    int           offset[16384];          // Segment offset position
    int           nlevel;                 // Number of quantized levels (2/4/16/256)
    unsigned char *k5head_ptr;            // Pointer to the K5 header
    struct        SHM_PARAM *param_ptr;   // Pointer to the Shared Param
    struct        sembuf sops;            // Semaphore for data access
    unsigned char *k5data_ptr;            // Pointer to shared K5 data
    float         *aspec_ptr;
    float         *xspec_ptr;             // Pointer to 1-sec-integrated Power Spectrum
    FILE          *file_ptr[3];           // File Pointer to write
    FILE          *power_ptr[2];          // Power File Pointer to write
    char          fname_pre[16];
    unsigned int  bitDist[1024];
    double        param[2], param_err[2]; // Gaussian parameters derived from bit distribution

    dim3          Dg, Db(512,1, 1);       // Grid and Block size
    unsigned char *cuk5data_ptr;          // Pointer to K5 data
    hipfftHandle   cufft_plan;             // 1-D FFT Plan, to be used in cufft
    hipfftReal     *cuRealData;            // Time-beased data before FFT, every IF, every segment
    hipfftComplex  *cuSpecData;            // FFTed spectrum, every IF, every segment
    float         *cuPowerSpec;           // (autocorrelation) Power Spectrum
    // float2        *cuASpec;
    float2        *cuXSpec;

    // Access to the SHARED MEMORY
    shrd_param_id = shmget(SHM_PARAM_KEY, sizeof(struct SHM_PARAM), 0444);
    param_ptr  = (struct SHM_PARAM *)shmat(shrd_param_id, NULL, 0);
    k5data_ptr = (unsigned char *)shmat(param_ptr->shrd_k5data_id, NULL, SHM_RDONLY);
    aspec_ptr  = (float *)shmat(param_ptr->shrd_aspec_id, NULL, 0);
    xspec_ptr  = (float *)shmat(param_ptr->shrd_xspec_id, NULL, 0);
    k5head_ptr = (unsigned char *)shmat(param_ptr->shrd_k5head_id, NULL, SHM_RDONLY);

    // Prepare for CuFFT
    hipMalloc((void **)&cuk5data_ptr, MAX_SAMPLE_BUF);
    hipMalloc((void **)&cuRealData, Nif* NsegPart* NFFT * sizeof(hipfftReal) );
    hipMalloc((void **)&cuSpecData, Nif* NsegPart* NFFTC* sizeof(hipfftComplex) );
    hipMalloc((void **)&cuPowerSpec, Nif* NFFT2* sizeof(float));
    hipMalloc((void **)&cuXSpec, 2* NFFT2* sizeof(float2));

    if(hipGetLastError() != hipSuccess){
        fprintf(stderr, "Cuda Error : Failed to allocate memory.\n");
        return(-1);
    }
    printf("cuda_fft_xspec: NFFT = %d, Nif = %d, NsegPart = %d\n", NFFT, Nif, NsegPart);

    if(hipfftPlan1d(&cufft_plan, NFFT, HIPFFT_R2C, Nif* NsegPart ) != HIPFFT_SUCCESS){
        fprintf(stderr, "Cuda Error : Failed to create plan.\n");
        return(-1);
    }

    // Parameters for S-part format
    // printf("NsegPart = %d\n", NsegPart);
    segment_offset(param_ptr, offset);
    // printf("segoff: %d\n", segment_offset);
    nlevel = 0x01<<(param_ptr->qbit); // Number of levels = 2^qbit

    // K5 Header and Data
    param_ptr->current_rec = 0;
    setvbuf(stdout, (char *)NULL, _IONBF, 0);   // Disable stdout cache

    while(param_ptr->validity & ACTIVE){
        if(param_ptr->validity & (FINISH + ABSFIN)){
            break;
        }
        hipMemset(cuPowerSpec, 0, Nif* NFFT2* sizeof(float));  // Clear Power Spec
        hipMemset(cuXSpec, 0, 2* NFFT2* sizeof(float2));       // Clear Power Spec

        // UTC in the K5 header
        while(k5utc(k5head_ptr, param_ptr) == 0){
            printf("%d\n", k5head_ptr[4]);
            usleep(100000);
        }

        // Open output files
        if(param_ptr->current_rec == 0){
            sprintf(fname_pre, "%04d%03d%02d%02d%02d", param_ptr->year, param_ptr->doy, param_ptr->hour, param_ptr->min, param_ptr->sec );
            for(index=0; index<Nif; index++){
                fileRecOpen(param_ptr, index, (A00_REC << index), fname_pre, "A", file_ptr);  // Autocorr
                fileRecOpen(param_ptr, index, (P00_REC << index), fname_pre, "P", power_ptr); // Bitpower
            }
            /* for(index=0; index<Nif/2; index++){
                fileRecOpen(param_ptr, index, (C00_REC << index), fname_pre, "C", &file_ptr[Nif]);  // Crosscorr
            } */
            fileRecOpen(param_ptr, Nif + 1, C00_REC, fname_pre, "C", &file_ptr[Nif + 1]);     // Crosscorr
        }

        // Loop for half-sec period
        memset(bitDist, 0, sizeof(bitDist));
        for(part_index=0; part_index<PARTNUM; part_index ++){
            // Wait for the first half in the S-part
            sops.sem_num = (ushort)(2* part_index);
            sops.sem_op = (short)-1;
            sops.sem_flg = (short)0;
            semop(param_ptr->sem_data_id, &sops, 1);

            // Move K5-sample data onto GPU memory
            hipMemcpy(&cuk5data_ptr[part_index* HALFBUF], &k5data_ptr[part_index* HALFBUF], HALFBUF, hipMemcpyHostToDevice);

            // Segment Format and Bit Distribution
            hipDeviceSynchronize();
            Dg.x=NFFT/512; Dg.y=1; Dg.z=1;
            if(nlevel == 256){
                for(index=0; index < NsegPart; index ++){
                    seg_index = part_index* NsegPart + index;
                    segform8bit<<<Dg, Db>>>( &cuk5data_ptr[4* offset[seg_index]], &cuRealData[index* Nif* NFFT], NFFT);
                }
                bitDist8( HALFBUF, &k5data_ptr[part_index* HALFBUF], bitDist);
            }
            else{
                for(index=0; index < NsegPart; index ++){
                    seg_index = part_index* NsegPart + index;
                    segform4bit<<<Dg, Db>>>( &cuk5data_ptr[2* offset[seg_index]], &cuRealData[index* Nif* NFFT], NFFT);
                }
                bitDist4( HALFBUF, &k5data_ptr[part_index* HALFBUF], bitDist);
            }

            // FFT Real -> Complex spectrum
            hipDeviceSynchronize();
            hipfftExecR2C(cufft_plan, cuRealData, cuSpecData);	// FFT Time -> Freq
            hipDeviceSynchronize();

            // Auto Correlation
            Dg.x= NFFTC/512; Dg.y=1; Dg.z=1;
            for(seg_index=0; seg_index<NsegPart; seg_index++){
                for(index=0; index<Nif; index++){
                    accumPowerSpec<<<Dg, Db>>>( &cuSpecData[(seg_index* Nif + index)* NFFTC], &cuPowerSpec[index* NFFT2],  NFFT2);
                }
            }

            // Cross Correlation
            for(seg_index=0; seg_index<NsegPart; seg_index++){
                accumCrossSpec<<<Dg, Db>>>(
                    &cuSpecData[(seg_index* Nif)* NFFTC],
                    &cuSpecData[(seg_index* Nif + 1)* NFFTC],
                    cuXSpec,
                    NFFT2
                );
                /* accumCrossSpec<<<Dg, Db>>>(
                        &cuSpecData[(seg_index* Nif + 2)*NFFTC],
                        &cuSpecData[(seg_index* Nif + 3)*NFFTC],
                        &cuXSpec[NFFT2],
                        NFFT2
                ); */
            }
        }  // End of part loop

        Dg.x = Nif* NFFT2/512; Dg.y = 1; Dg.z = 1;
        scalePowerSpec<<<Dg, Db>>>(cuPowerSpec, SCALEFACT, Nif* NFFT2);
        scaleCrossSpec<<<Dg, Db>>>(cuXSpec, SCALEFACT, NFFT2);

        // Dump cross spectra to shared memory
        hipMemcpy(aspec_ptr, cuPowerSpec, Nif* NFFT2* sizeof(float), hipMemcpyDeviceToHost);
        for(index=0; index<Nif; index++){
            if(file_ptr[index] != NULL){
                fwrite(&aspec_ptr[index* NFFT2], sizeof(float), NFFT2, file_ptr[index]); // Save Pspec
            }
            if(power_ptr[index] != NULL){
                fwrite(&bitDist[index* nlevel], sizeof(int), nlevel, power_ptr[index]);  // Save Bitdist
            }

            // Total Power calculation
            gaussBit( nlevel, &bitDist[nlevel* index], param, param_err );
            param_ptr->power[index] = 1.0/(param[0]* param[0]);
        }
        hipMemcpy(xspec_ptr, cuXSpec, 2* NFFT2* sizeof(float2), hipMemcpyDeviceToHost);

        /* for(index=0; index<Nif/2; index++){
            if(file_ptr[Nif + index] != NULL){
                fwrite(&xspec_ptr[(index * 2)* NFFT2], sizeof(float2), NFFT2, file_ptr[Nif + index]); // Save Xspec
            }
        } */
        if(file_ptr[Nif +  1] != NULL){
            fwrite(&xspec_ptr[NFFT2], sizeof(float2), NFFT2, file_ptr[Nif + 1]);  // Save Xspec
        }

        // Refresh output data file
        if(param_ptr->current_rec == MAX_FILE_REC - 1){
            for(index=0; index<Nif+1; index++){
                if(file_ptr[index] != NULL){
                    fclose(file_ptr[index]);
                }
            }
            for(index=0; index<Nif; index++){
                if( power_ptr[index] != NULL){
                    fclose(power_ptr[index]);
                }
            }
            param_ptr->current_rec = 0;
        }
        else{
            param_ptr->current_rec ++;
        }

        sops.sem_num = (ushort)SEM_FX; sops.sem_op = (short)1; sops.sem_flg = (short)0; semop( param_ptr->sem_data_id, &sops, 1);
        sops.sem_num = (ushort)SEM_POWER; sops.sem_op = (short)1; sops.sem_flg = (short)0; semop( param_ptr->sem_data_id, &sops, 1);
        printf("%04d %03d UT %02d:%02d:%02d Rec %d / %d -- Power = %8.3f %8.3f %8.3f %8.3f\n", param_ptr->year, param_ptr->doy, param_ptr->hour, param_ptr->min, param_ptr->sec, param_ptr->current_rec, param_ptr->integ_rec, param_ptr->power[0], param_ptr->power[1], param_ptr->power[2], param_ptr->power[3]);
    } // End of 1-sec loop


    // RELEASE the SHM
    for(index=0; index<Nif+1; index++){
        if(file_ptr[index] != NULL){
            fclose(file_ptr[index]);
        }
    }
    for(index=0; index<Nif; index++){
        if( power_ptr[index] != NULL){
            fclose(power_ptr[index]);
        }
    }
    hipfftDestroy(cufft_plan);
    hipFree(cuk5data_ptr);
    hipFree(cuRealData);
    hipFree(cuSpecData);
    hipFree(cuPowerSpec);
    hipFree(cuXSpec);

    return(0);
}
